#include "hip/hip_runtime.h"
#include "swnlmcuda.cuh"
#include "swilk.cuh"

#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace cv;

template <typename T>
__global__ void kernel(const T *in, const double *a, double *sumWeights, double *avg, int rows, int cols, int searchRadius, int neighborRadius, double sigma)
{
    const size_t searchDiam = 2 * searchRadius + 1;

    const size_t threadNum = blockIdx.x * blockDim.x + threadIdx.x;

    const size_t row = threadNum / (cols *searchDiam * searchDiam); 
    const size_t col = (threadNum / (searchDiam * searchDiam)) % cols;

    const size_t sRow = threadNum % (searchDiam * searchDiam) / searchDiam; // 0 <= sRow < 21 for searchRadius =10
    const size_t sCol = threadNum % searchDiam;

    const size_t padding = searchRadius + neighborRadius;

    const size_t paddedRow = row + padding;
    const size_t paddedCol = col + padding;
    const size_t paddedSRow = paddedRow + sRow - searchRadius;
    const size_t paddedSCol = paddedCol + sCol - searchRadius;

    const size_t inCols = cols + 2 * padding;

    extern __shared__ double diffArr[]; // Shared memory space that is sliced and used for each thread locally

    bool accepted = false;
    double w = 0;
    double res = 0;

    if (paddedSRow == paddedRow && paddedSCol == paddedCol)
    { // center pixel is skipped
        w = 1;
        res = 1 * in[paddedRow * inCols + paddedCol];
        accepted = true;
    }
    else
    {
        const int numNeighbors = (neighborRadius * 2 + 1) * (neighborRadius * 2 + 1);

        double *diff = diffArr + numNeighbors * (threadIdx.x);

        const int neighborDiam = neighborRadius * 2 + 1;
        for (int y = 0; y < neighborDiam; y++)
        {
            for (int x = 0; x < neighborDiam; x++)
            {
                const int diffIdx = y * neighborDiam + x;

                const int iNghbrIdx = inCols * (paddedRow + y - neighborRadius) + paddedCol + x - neighborRadius;
                const int jNghbrIdx = inCols * (paddedSRow + y - neighborRadius) + paddedSCol + x - neighborRadius;

                diff[diffIdx] = (in[iNghbrIdx] - in[jNghbrIdx]) / (sqrt(2.0) * sigma);
            }
        }

        bool hypothesis;
        ShapiroWilk::test(diff, a, numNeighbors, w, hypothesis);

        double mean = 0;
        for (int i = 0; i < numNeighbors; i++)
        {
            mean += diff[i];
        }
        mean /= numNeighbors;

        double stddev = 0;
        for (int i = 0; i < numNeighbors; i++)
        {
            stddev += (diff[i] - mean) * (diff[i] - mean);
        }
        stddev /= numNeighbors;
        stddev = sqrt(stddev);

        const double stderror = stddev / neighborDiam; // Neighborhoods are square, thus sqrt(n) observations is number of rows

        if (stderror > mean && mean > -stderror &&
            (1 + stderror > stddev && stddev > 1 - stderror) &&
            hypothesis) // Fail to reject Null hypothesis that it is normally distributed
        {
            res = w * in[paddedSRow * inCols + paddedSCol];
            accepted = true;
        }
    }
    if (accepted)
    {
        atomicAdd(&sumWeights[row * cols + col], w);
        atomicAdd(&avg[row * cols + col], res);
    }
}

//
template <typename T>
__global__ void denoiseStep(double *sumWeights, double *avg, T *out, const int rows, const int cols)
{
    const int threadNum = blockIdx.x * blockDim.x + threadIdx.x;

    out[threadNum] = avg[threadNum] / sumWeights[threadNum];
}

template void swnlmcuda(const Mat &noisyImage, Mat &denoised, const short sigma, const int searchRadius, const int neighborRadius);
template void swnlmcuda(const Mat &noisyImage, Mat &denoised, const float sigma, const int searchRadius, const int neighborRadius);
template void swnlmcuda(const Mat &noisyImage, Mat &denoised, const double sigma, const int searchRadius, const int neighborRadius);

template <typename T>
void swnlmcuda(const Mat &noisyImage, Mat &denoised, const T sigma, const int searchRadius, const int neighborRadius)
{
    assert(noisyImage.type() == cv::DataType<T>::type);
    assert(noisyImage.dims == 2);

    const int rows = noisyImage.rows;
    const int cols = noisyImage.cols;

    // Pad the edges with a reflection of the outer pixels.
    const int padding = searchRadius + neighborRadius;
    Mat paddedImage;
    copyMakeBorder(noisyImage, paddedImage, padding, padding, padding, padding, BORDER_REFLECT);

    const int paddedFlat[] = {(int)paddedImage.total()};
    paddedImage = paddedImage.reshape(0, 1, paddedFlat);
    T *h_in = (T *)paddedImage.data;

    const int numNeighbors = (neighborRadius * 2 + 1) * (neighborRadius * 2 + 1);
    vector<double> h_a(numNeighbors + 1);
    ShapiroWilk::setup(h_a.data(), numNeighbors);

    const int searchDiam = 2 * searchRadius + 1;

    const int totalThreads = rows * cols * searchDiam * searchDiam;
    const int threadsPerBlock = 32; // Arbitrarily chosen, needs more experimentation.
    const int numBlocks = ceil((double)totalThreads / threadsPerBlock);

    dim3 blocks(numBlocks);
    dim3 threads(threadsPerBlock);

    T *d_in, *d_out;
    double *d_a;
    const size_t inSize = paddedImage.total() * paddedImage.channels() * paddedImage.elemSize();
    hipMalloc(&d_in, inSize);
    assert(d_in != NULL);
    hipMemcpyAsync(d_in, h_in, inSize, hipMemcpyHostToDevice);

    hipMalloc(&d_a, numNeighbors * sizeof(double));
    assert(d_a != NULL);
    hipMemcpyAsync(d_a, h_a.data(), numNeighbors * sizeof(double), hipMemcpyHostToDevice);

    double *d_sumWeights, *d_avg;
    hipMalloc(&d_sumWeights, rows * cols * sizeof(double));
    assert(d_sumWeights != NULL);
    hipMemset(d_sumWeights, 0, rows * cols * sizeof(double));

    hipMalloc(&d_avg, rows * cols * sizeof(double));
    assert(d_avg != NULL);
    hipMemset(d_avg, 0, rows * cols * sizeof(double));

    // Allocate output array
    const int flatShape[] = {rows * cols};
    denoised.create(1, flatShape, noisyImage.type());
    T *h_out = (T *)denoised.data;

    const size_t sharedMemSize = numNeighbors * threadsPerBlock * sizeof(double);
    hipDeviceSynchronize();

    kernel<T><<<blocks, threads, sharedMemSize>>>(d_in, d_a, d_sumWeights, d_avg, rows, cols, searchRadius, neighborRadius, sigma);

    const size_t outSize = denoised.total() * denoised.channels() * denoised.elemSize();
    hipMalloc(&d_out, outSize);
    assert(d_out != NULL);

    dim3 denoiseBlocks(ceil((double)rows * cols / threadsPerBlock));
    dim3 denoiseThreads(threadsPerBlock);

    hipDeviceSynchronize();

    denoiseStep<T><<<denoiseBlocks, denoiseThreads>>>(d_sumWeights, d_avg, d_out, rows, cols);

    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, outSize, hipMemcpyDeviceToHost);

    const int shape[] = {rows, cols};
    denoised = denoised.reshape(0, 2, shape);

    hipFree(d_in);
    hipFree(d_a);
    hipFree(d_sumWeights);
    hipFree(d_avg);
    hipFree(d_out);

    hipProfilerStop();
}